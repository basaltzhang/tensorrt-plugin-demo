#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "plugin.h"
#include "gatherNMSOutputs.h"
#include <vector>

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void gatherNMSOutputs_kernel(
        const bool shareLocation,
        const int numImages,
        const int numPredsPerClass,
        const int numClasses,
        const int topK,
        const int keepTopK,
        const int* indices,
        const T_SCORE* scores,
        const T_BBOX* bboxData,
        int* nmsedIndices
        )
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x;
         i < numImages * keepTopK;
         i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId];
        if (index == -1) {
            nmsedIndices[i] = -1;
        } else {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            nmsedIndices[i] = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + bboxOffset);
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSOutputs_gpu(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* nmsedIndices
    )
{
    hipMemsetAsync(nmsedIndices, -1, keepTopK * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
                                                                           numClasses, topK, keepTopK,
                                                                           (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData,
                                                                           (int*) nmsedIndices
                                                                            );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t,
                               const bool,
                               const int,
                               const int,
                               const int,
                               const int,
                               const int,
                               const void*,
                               const void*,
                               const void*,
                               void*
                               );
struct nmsOutLaunchConfig
{
    DataType t_bbox;
    DataType t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DataType t_bbox, DataType t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::vector<nmsOutLaunchConfig> nmsOutFuncVec;

bool nonMaxSuppressionOutputInit()
{
    nmsOutFuncVec.push_back(nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT,
                                         gatherNMSOutputs_gpu<float, float>));
    return true;
}

static bool initialized = nonMaxSuppressionOutputInit();

//}}}

pluginStatus_t gatherNMSOutputs(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const DataType DT_BBOX,
    const DataType DT_SCORE,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* nmsedIndices
    )
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutFuncVec.size(); ++i)
    {
        if (lc == nmsOutFuncVec[i])
        {
            DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i); return nmsOutFuncVec[i].function(stream,
                                          shareLocation,
                                          numImages,
                                          numPredsPerClass,
                                          numClasses,
                                          topK,
                                          keepTopK,
                                          indices,
                                          scores,
                                          bboxData,
                                          nmsedIndices
                                          );
        }
    }
    return STATUS_BAD_PARAM;
}
